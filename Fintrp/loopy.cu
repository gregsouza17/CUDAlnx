#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


void free_data(double ***data, int xlen, int ylen);


extern __shared__ double cache[];

__global__ void kernel(int *Ss, int *Nn, int *mask, double *xyz,
		       double *cost){

  //block idx in 0, S
  //thread idx X 0,S and thread Idx Y 0,N
  //mask (N*k + j), cost( i1) xyz (N*S*k + S*j + i2)
  //cache =[threadIdx.x + threadIdx.y*blockDim.x]

  double temp=0;
  long int i1,i2,j, joffset=blockDim.y,i2offset=blockDim.x;
  int k, N=*Nn, S = *Ss, cacheIndex, cIndexMax;
  i1 = blockIdx.x; i2 = threadIdx.x;

  cacheIndex = threadIdx.x + threadIdx.y*blockDim.x;
  cIndexMax = blockDim.x*blockDim.y;


  while(i2<S){
   
    j = threadIdx.y;
    while(j<N){
      if (i1!=i2){ 
	// temp=0; 
	for(k=0; k<3; k++){
      
	  if( mask[k*N+j] ){ 
	
	//	temp+=1; 

	    //  printf("%f \t", xyz[i1+S*(j+N*k)]);
	    temp+=
	      (xyz[i1+S*(j+N*k)] - xyz[i2+S*(j+N*k)])* 
	      (xyz[i1+S*(j+N*k)] - xyz[i2+S*(j+N*k)]) ; 
	
	  } //if mask 

	} //k

      } //if i1!=i2
  
   
      __syncthreads();
      j+=joffset;
  } //while j<N;
    __syncthreads();
   i2+=i2offset;
  } //while i2<S

  __syncthreads();
   cache[cacheIndex]+= temp;


 

 
  //Somar todos os indices do cache aqui
  //ofset separado

  
  int i = cIndexMax/2;
  while (i != 0) {
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex + i];
    __syncthreads();
    i /= 2;
    
  }


  __syncthreads();

  cost[i1] = cache[0];
 
}

int main()
{

  //Max double array length: 523268
  // Max float array length: 1046537


  //Initializing
  long int N = 32, S = 32, sizexyz = N*S*3; 
  double xyz[3][N][S]; 
  double *linxyz;  
  double cost[S], soma;
  int mask[3][N]={0}, linmask[3*N];
  long int i1,i2;
  long int j=0,k=0;
 
  linxyz = (double *)malloc(sizexyz*sizeof(double));
  
  

  //mask
    for(k=0; k<3; k++){
      for(j=0; j<N; j++){
	mask[k][j] = 1;
	if(j%(k+1)==0)
	  mask[k][j] = 1;	
      }
    }
   
    for(k=0; k<3; k++){
      for(j=0; j<N; j++){
	linmask[j+N*k] = mask[k][j];
      }
    }


    //mask

    for(k=0; k<3; k++){
      for(j=0; j<N;j++){
	for(i1=0; i1<S; i1++){
	  xyz[k][j][i1] = 0.0001*i1;
	  linxyz[i1+S*j + S*N*k] = xyz[k][j][i1];
	  // printf("%f \n", linxyz[i1+S*j+S*N*k]);
	}
      }
    }


    //CPU
    soma = 0;

  for (i1 = 0  ; i1 < S ; ++i1) {    
    for (i2 = 0;   i2< S ; ++i2) {
        if(i1!=i2){
      	soma = 0;
      	for(j=0;   j<N;   j++){
	 
      	  for(k=0;   k<3;    k++){

	    if( mask[k][j] ){
	     
	      soma+=
		(linxyz[k*S*N+j*S +i1] -linxyz[k*S*N+j*S +i2])*
		(xyz[k][j][i1] -linxyz[k*S*N+j*S +i2]);
	      // }
	    }
	  } //for k
	
	} //for j
       	cost[i1]+=soma;
	} //for if
    } //for i2
  } //for i1

  //GPU

  int *devN, *devS;
  
  hipMalloc((void **)&devN, sizeof(int));
  hipMalloc((void **)&devS, sizeof(int));

  hipMemcpy(devN, &N, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devS, &S, sizeof(int), hipMemcpyHostToDevice);

  int *dmask;
  hipMalloc((void **)&dmask, sizeof(linmask));

  hipMemcpy(dmask, linmask, sizeof(linmask), hipMemcpyHostToDevice);

  double *d_xyz, *d_cost, cost2[S]={0};
  hipMalloc((void **)&d_xyz,sizexyz*sizeof(double) );
  hipMalloc((void **)&d_cost, sizeof(cost));

  printf("%f \n" ,linxyz[17]);
  hipMemcpy(d_xyz, linxyz, sizexyz*sizeof(double), hipMemcpyHostToDevice);

  int threadX=16, threadY=16, cacheSize;

 
  dim3 threads(threadX,threadY);
  cacheSize = threadX*threadY;

  kernel<<<S, threads, cacheSize*sizeof(double)>>>(devS,devN, dmask, d_xyz,
						   d_cost);

  hipMemcpy(cost2, d_cost, sizeof(cost2), hipMemcpyDeviceToHost);
  

  for(i1=0; i1<S; i1+=10){
     printf("i1: %ld cost: %f dcost: %f\n", i1,cost[i1], cost2[i1]);
   }
  //printf("i1: %d cost: %f dcost: %f\n", 30,cost[30], cost2[30]);
  
  hipFree(devN); hipFree(devS); hipFree(dmask);
  hipFree(d_xyz); hipFree(d_cost); free(linxyz);

  //free_data(xyz, 3, N);
  

  return 0;
}


void free_data(double ***data, int xlen, int ylen)
{
    size_t i, j;

    for (i=0; i < xlen; ++i) {
        if (data[i] != NULL) {
            for (j=0; j < ylen; ++j)
                free(data[i][j]);
            free(data[i]);
        }
    }
    free(data);
}
