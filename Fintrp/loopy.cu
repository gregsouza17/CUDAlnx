#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>




extern __shared__ double cache[];

__global__ void kernel(int *Ss, int *Nn, int *mask, double *xyz,
		       double *cost){

  //block idx in 0, S
  //thread idx X 0,S and thread Idx Y 0,N
  //mask (N*k + j), cost( i1) xyz (N*S*k + S*j + i2)
  //cache =[threadIdx.x + threadIdx.y*blockDim.x]

  double temp=0;
  long int i1,i2,j;
  int k, N=*Nn, S = *Ss, cacheIndex, cIndexMax;
  i1 = blockIdx.x; i2 = threadIdx.x; j = threadIdx.y;

  cacheIndex = threadIdx.x + threadIdx.y*blockDim.x;
  cIndexMax = blockDim.x*blockDim.y;


   if (i1!=i2){ 
     temp=0; 
     for(k=0; k<3; k++){
      
      if(true){ 
	
	//	temp+=1; 

	temp+=
   	   (xyz[i1+S*(j+N*k)] - xyz[i2+S*(j+N*k)])* 
  	   (xyz[i1+S*(j+N*k)] - xyz[i2+S*(j+N*k)]); 
	
      } //if mask 
        __syncthreads();   
     } //k
     // __syncthreads();
   } //if i1!=i2 

   __syncthreads();
  
  cache[cacheIndex] = temp;

 

 
  //Somar todos os indices do cache aqui
  //ofset separado

  
  int i = cIndexMax/2;
  while (i != 0) {
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex + i];
    __syncthreads();
    i /= 2;
    
  }


  __syncthreads();

  if(i1==30 && (j+i2)==0)
      printf("%f \n", cache[0]);
  
  cost[i1] = cache[0];
 
}

int main()
{
  //Initializing
  int N = 32, S = 32;
  double xyz[3][N][S], linxyz[3*N*S],soma;
  double cost[S];
  int mask[3][N]={0}, linmask[3*N];
  long int i1,i2;
  long int j=0,k=0;

  //mask
    for(k=0; k<3; k++){
      for(j=0; j<N; j++){
	mask[k][j] = 1;
	if(j%(k+1)==0)
	  mask[k][j] = 1;	
      }
    }
   
    for(k=0; k<3; k++){
      for(j=0; j<N; j++){
	linmask[j+N*k] = mask[k][j];
      }
    }
    //mask

    for(k=0; k<3; k++){
      for(j=0; j<N;j++){
	for(i1=0; i1<S; i1++){
	  xyz[k][j][i1] = 0.01*i1;
	  linxyz[i1+S*j + S*N*k] = xyz[k][j][i1];
	}
      }
    }


    //CPU
    soma = 0;

  for (i1 = 0  ; i1 < S ; ++i1) {    
    for (i2 = 0;   i2< S ; ++i2) {
        if(i1!=i2){
      	soma = 0;
      	for(j=0;   j<N;   j++){
	  
      	  for(k=0;   k<3;    k++){

	    if( linmask[k*N+j] ){
	      soma+=
		(linxyz[k*N*S+j*S+i1] - linxyz[k*N*S+j*S+i2])*
		(linxyz[k*N*S+j*S+i1] - linxyz[k*N*S+j*S+i2]);
	      // }
	    }
	  } //for k
	} //for j
       	cost[i1]+=soma;
	  } //for if
    } //for i2
  } //for i1

  //GPU

  int *devN, *devS;
  
  hipMalloc((void **)&devN, sizeof(int));
  hipMalloc((void **)&devS, sizeof(int));

  hipMemcpy(devN, &N, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devS, &S, sizeof(int), hipMemcpyHostToDevice);

  int *dmask;
  hipMalloc((void **)&dmask, sizeof(linmask));

  hipMemcpy(dmask, linmask, sizeof(linmask), hipMemcpyHostToDevice);

  double *d_xyz, *d_cost, cost2[S]={0};
  hipMalloc((void **)&d_xyz, sizeof(linxyz));
  hipMalloc((void **)&d_cost, sizeof(cost));

  hipMemcpy(d_xyz, linxyz, sizeof(linxyz), hipMemcpyHostToDevice);

  int threadX=S, threadY=N, cacheSize;

 
  dim3 threads(threadX,threadY);
  cacheSize = threadX*threadY;

  kernel<<<S, threads, cacheSize*sizeof(double)>>>(devS,devN, dmask, d_xyz,
						   d_cost);

  hipMemcpy(cost2, d_cost, sizeof(cost2), hipMemcpyDeviceToHost);
  

  for(i1=0; i1<N; i1+=2){
   printf("i1: %ld cost: %f dcost: %f\n", i1,cost[i1], cost2[i1]);
   }
  //printf("i1: %d cost: %f dcost: %f\n", 30,cost[30], cost2[30]);
  
  hipFree(devN); hipFree(devS); hipFree(dmask);
  hipFree(d_xyz); hipFree(d_cost);
  

  return 0;
}
