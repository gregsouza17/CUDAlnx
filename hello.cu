#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int *v1, int *v2, int *sol) {
	*sol = *v1 + *v2;
	printf("Hello Cuda!\n");
}


int main(void) {
	int v1, v2, sol; //Host copies of the values
	int *d_v1, *d_v2, *d_sol; //int vector for v1,v2,v3 in the device
	int size = sizeof(int);

	//Allocating space in the device
	hipMalloc((void **)&d_v1, size);
	hipMalloc((void **)&d_v2, size);
	hipMalloc((void **)&d_sol, size);

	//setup input
	v1 = 17;
	v2 = 13;

	//Input values to device
	hipMemcpy(d_v1, &v1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_v2, &v2, size, hipMemcpyHostToDevice);
	hipMemcpy(d_sol, &sol, size, hipMemcpyHostToDevice);

	//Lauch add() in device
	add << <1, 1 >> >(d_v1, d_v2, d_sol);

	//Copy result from device to host
	hipMemcpy(&sol, d_sol, size, hipMemcpyDeviceToHost);

	//Cleanup
	printf(" Hello %d \n", sol);
	hipFree(d_v1);
	hipFree(d_v2);
	hipFree(d_sol);

	return 0;


}
